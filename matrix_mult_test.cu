#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Include your custom kernel(s)
#include "Kernel1.cu"
#include "Kernel2.cu"
#include "Kernel3.cu"
#include "Kernel4.cu"
#include "Kernel5.cu"
#include "Kernel6.cu"

void initializeMatrix(float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

//some useful variables 
const int M = 1024;
const int N = 1024;
const int K = 1024;

hipSetDevice(0);
hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(float) * M * K);
    hipMalloc((void**)&d_B, sizeof(float) * K * N);
    hipMalloc((void**)&d_C, sizeof(float) * M * N);

    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    initializeMatrix(h_A, M * K);
    initializeMatrix(h_B, K * N);

    hipMemcpy(d_A, h_A, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float alpha = 1.0f;
    float beta = 0.0f;

    std::vector<std::function<void()>> kernelExecutions = {
        //CUBLAS Default 
        [&]() {
            hipEventRecord(start);
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            double seconds = milliseconds / 1000.0;
            double flops = 2.0 * M * N * K / seconds;
            std::cout << "FLOPs for CUBLAS: " << flops << std::endl;
        },
        // Kernel 1
        [&]() {
            dim3 blockSize1(16, 16);
            dim3 gridSize1(CEIL_DIV(M, blockSize1.x), CEIL_DIV(N, blockSize1.y));
            hipEventRecord(start);
            sgemm_shared_mem_block<16><<<gridSize1, blockSize1>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds1 = 0;
            hipEventElapsedTime(&milliseconds1, start, stop);
            double seconds1 = milliseconds1 / 1000.0;
            double flops1 = 2.0 * M * N * K / seconds1;
            std::cout << "FLOPs for Custom Kernel1: " << flops1 << std::endl;
        },
        // Kernel 2
        // Repeat the above structure for each kernel with their specific configurations
        // For example:
        [&]() {
            const int BM = 64; // arbitrary value for BM
            const int BN = 64; // arbitrary value for BN
            const int BK = 8; // arbitrary value for BK
            const int TM = 8;  // arbitrary value for TM

            dim3 blockSize1(16, 16);
            dim3 gridSize1(CEIL_DIV(M, blockSize1.x), CEIL_DIV(N, blockSize1.y));
            hipEventRecord(start);
            sgemm_multi_entry_per_thread(int M, int N, int K, float alpha,
                                             const float *A, const float *B, float beta,
                                             float *C)
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds2 = 0;
            hipEventElapsedTime(&milliseconds2, start, stop);
            double seconds2 = milliseconds2 / 1000.0;
            double flops2 = 2.0 * M * N * K / seconds2;
            std::cout << "FLOPs for Custom Kernel1: " << flops1 << std::endl;
        },
        // Kernel 3 and beyond will be soon 
    };

    for (auto& exec : kernelExecutions) {
        exec();
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}






