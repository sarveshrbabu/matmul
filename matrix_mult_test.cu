#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Include your custom kernel(s)
#include "Kernel1.cu"
#include "Kernel2.cu"
#include "Kernel3.cu"
#include "Kernel4.cu"
#include "Kernel5.cu"
#include "Kernel6.cu"

void initializeMatrix(float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

void printMatrix(const float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Matrix dimensions
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    // Initialize CUDA and CUBLAS
    hipSetDevice(0);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate device memory for matrices
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(float) * M * K);
    hipMalloc((void**)&d_B, sizeof(float) * K * N);
    hipMalloc((void**)&d_C, sizeof(float) * M * N);

    // Allocate host memory for matrices
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    // Initialize matrices with random values
    initializeMatrix(h_A, M * K);
    initializeMatrix(h_B, K * N);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice);

    // Test custom kernel
    dim3 blockSize(16, 16);
    dim3 gridSize(CEIL_DIV(M, blockSize.x), CEIL_DIV(N, blockSize.y));

    float alpha = 1.0;
    float beta = 0.0;

    // Test custom kernel1
    clock_t start_custom1 = clock();
    sgemm_shared_mem_block<16><<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom1 = clock();

    // Copy result back to host for kernel1
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Test custom kernel2
    clock_t start_custom2 = clock();
    sgemm_shared_mem_block<32><<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom2 = clock();

    // Copy result back to host for kernel2
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    /*
    // Test custom kernel3
    clock_t start_custom3 = clock();
    sgemm2DBlocktiling<<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom3 = clock();

    // Copy result back to host for kernel3
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Calculate FLOPs for custom kernel3
    double flops_custom3 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom3 - start_custom3) * CLOCKS_PER_SEC;

    // Print FLOPs for kernel3
    std::cout << "FLOPs for Custom Kernel3: " << flops_custom3 << std::endl;

    // Test custom kernel4
    clock_t start_custom4 = clock();
    sgemm_warpshuffling<<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom4 = clock();

    // Copy result back to host for kernel4
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Calculate FLOPs for custom kernel4
    double flops_custom4 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom4 - start_custom4) * CLOCKS_PER_SEC;

    // Print FLOPs for kernel4
    std::cout << "FLOPs for Custom Kernel4: " << flops_custom4 << std::endl;

    // Test custom kernel5
    clock_t start_custom5 = clock();
    sgemm_vectorized<<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom5 = clock();

    // Copy result back to host for kernel5
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Calculate FLOPs for custom kernel5
    double flops_custom5 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom5 - start_custom5) * CLOCKS_PER_SEC;

    // Print FLOPs for kernel5
    std::cout << "FLOPs for Custom Kernel5: " << flops_custom5 << std::endl;

    // Test custom kernel6
    clock_t start_custom6 = clock();
    sgemm2DBlockTilingAutotuned<<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipDeviceSynchronize();
    clock_t end_custom6 = clock();

    // Copy result back to host for kernel6
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Calculate FLOPs for custom kernel6
    double flops_custom6 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom6 - start_custom6) * CLOCKS_PER_SEC;

    // Print FLOPs for kernel6
    std::cout << "FLOPs for Custom Kernel6: " << flops_custom6 << std::endl;
    */
    // Test CUBLAS
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Calculate FLOPs for CUBLAS
    double flops_cublas = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / ((end_custom1 - start_custom1) + (end_custom2 - start_custom2)) * CLOCKS_PER_SEC;

    // Calculate FLOPs for custom kernel1
    double flops_custom1 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom1 - start_custom1) * CLOCKS_PER_SEC;

    // Calculate FLOPs for custom kernel2
    double flops_custom2 = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) / (end_custom2 - start_custom2) * CLOCKS_PER_SEC;

    // Print FLOPs for kernel1 and kernel2
    std::cout << "FLOPs for Custom Kernel1: " << flops_custom1 << std::endl;
    std::cout << "FLOPs for Custom Kernel2: " << flops_custom2 << std::endl;

    // Print proportion of FLOPs for kernel1 and kernel2
    std::cout << "Proportion of FLOPs for Kernel1: " << flops_custom1 / flops_cublas << std::endl;
    std::cout << "Proportion of FLOPs for Kernel2: " << flops_custom2 / flops_cublas << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}
